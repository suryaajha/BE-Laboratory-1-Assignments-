#include "hip/hip_runtime.h"
%%cu
#include<iostream>
#include<cstdio>
#include<math.h>
#include <time.h>

using namespace std;


__global__
void reduction_kernel(float *d_a, int expo){

	int tid = blockIdx.x * blockDim.x + threadIdx.x ;

	int i = tid * pow(2, expo) ; 
	int next = i + pow(2, expo) / 2 ; 

	d_a[i] = d_a[i] + d_a[next] ; 
}

__global__
void apply_kernel(float *d_a, float mean_val){
	int tid = blockIdx.x * blockDim.x + threadIdx.x ;

	d_a[tid] = (d_a[tid] - mean_val) * (d_a[tid] - mean_val) ; 
	printf("%d, %f\n", tid, d_a[tid]);
}

void printArr(float *a, int N){
	cout << "Array is" << endl ; 
	for (int i = 0; i < N; i++)
	{
		cout << a[i] << "\t" ;
	}
  	cout << endl ; 	
}

float trad_mean(float *a, int N){
	int sum = 0 ; 
	for (int i = 0; i < N; i++)
	{
		sum += a[i] ; 
	}
	return float(sum) / float(N) ;  
}

int foo(){

	int N = 8 ; 
	float a[N] , t[N]; 

	for (int i = 0; i < N; i++)
	{
		a[i] = i;
	}

	float t_min = trad_mean(a, N) ; 
	cout << "\nArithmetic Mean is " << t_min << endl; 
  	// printArr(a, N) ; 

	float *d_a;
	int bytes = sizeof(float) * N ; 

	hipMalloc(&d_a, bytes) ; 
	hipMemcpy(d_a, a, bytes, hipMemcpyHostToDevice) ; 

	int num_thread = N ; 
	int expo = 1 ; 
	while(num_thread != 0){
		num_thread = num_thread / 2 ; 
		reduction_kernel<<<1,num_thread>>>(d_a, expo) ;
		expo++ ;  

	}
	// reduction_kernel<<<1,N/2>>>(d_a) ; 

	// get everything into temp array t
	hipMemcpy(t, d_a, bytes, hipMemcpyDeviceToHost) ; 
	
	// reinit d_a with original a
	hipMemcpy(d_a, a, bytes, hipMemcpyHostToDevice) ; 

  	// printArr(a, N) ;
  	// printArr(a, N) ;
  	float c_min = float(t[0]) / float(N) ; 
  	cout << "\nArithmetic Mean  is " << c_min << endl ;

  	// just to do (xi minux mean) square
  	apply_kernel<<<1,N>>>(d_a, c_min) ; 

  	// Now Sum
	num_thread = N ; 
	expo = 1 ; 
	while(num_thread != 0){
		num_thread = num_thread / 2 ; 
		reduction_kernel<<<1,num_thread>>>(d_a, expo) ;
		expo++ ;  

	}

	// get everything into temp array t
	hipMemcpy(t, d_a, bytes, hipMemcpyDeviceToHost) ;
	float summation = t[0] ;
	cout << summation << endl ; 
	float variance = summation / float(N) ; 
	cout << variance << endl ; 

	cout << "Standard Deviation " << sqrt(variance) << endl ;

	hipFree(d_a) ; 

  	return t_min == c_min ; 

}

int main(){
  srand(time(0)) ; 
  
    int i = 0;
   	for (i = 0; i < 100; i++)
   	{
   		if (foo() == 0)
         break;
   	}
    cout << i << endl ;
    if(i == 100)
      cout << "Well Done" << endl ; 
    else
      cout << "Wrong" << endl ; 
	
	return 0 ; 
}