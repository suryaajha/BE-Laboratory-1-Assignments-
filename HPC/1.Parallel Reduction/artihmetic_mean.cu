#include "hip/hip_runtime.h"
%%cu
#include<iostream>
#include<cstdio>
#include<math.h>
#include <time.h>

using namespace std;


__global__
void reduction_kernel(int *d_a, int expo){

	int tid = blockIdx.x * blockDim.x + threadIdx.x ;

	int i = tid * pow(2, expo) ; 
	int next = i + pow(2, expo) / 2 ; 

	d_a[i] = d_a[i] + d_a[next] ; 
}

void printArr(int *a, int N){
	cout << "Array is" << endl ; 
	for (int i = 0; i < N; i++)
	{
		cout << a[i] << "\t" ;
	}
  	cout << endl ; 	
}

float trad_mean(int *a, int N){
	int sum = 0 ; 
	for (int i = 0; i < N; i++)
	{
		sum += a[i] ; 
	}
	return float(sum) / float(N) ;  
}

int foo(){

	int N = 8 ; 
	int a[N] ; 

	for (int i = 0; i < N; i++)
	{
		a[i] = i;
	}

	float t_min = trad_mean(a, N) ; 
	cout << "\nArithmetic Mean is " << t_min << endl; 
  	// printArr(a, N) ; 

	int *d_a;
	int bytes = sizeof(int) * N ; 

	hipMalloc(&d_a, bytes) ; 
	hipMemcpy(d_a, a, bytes, hipMemcpyHostToDevice) ; 

	int num_thread = N ; 
	int expo = 1 ; 
	while(num_thread != 0){
		num_thread = num_thread / 2 ; 
		reduction_kernel<<<1,num_thread>>>(d_a, expo) ;
		expo++ ;  

	}
	// reduction_kernel<<<1,N/2>>>(d_a) ; 

	hipMemcpy(a, d_a, bytes, hipMemcpyDeviceToHost) ; 

	hipFree(d_a) ; 

  	// printArr(a, N) ;
  	float c_min = float(a[0]) / float(N) ; 
  	cout << "\nArithmetic Mean  is " << c_min << endl ;

  	return t_min == c_min ; 

}

int main(){
  srand(time(0)) ; 
  
    int i = 0;
   	for (i = 0; i < 100; i++)
   	{
   		if (foo() == 0)
         break;
   	}
    cout << i << endl ;
    if(i == 100)
      cout << "Well Done" << endl ; 
    else
      cout << "Wrong" << endl ; 
	
	return 0 ; 
}