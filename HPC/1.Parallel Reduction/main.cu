#include "hip/hip_runtime.h"
%%cu
#include<iostream>
#include<cstdio>
#include<math.h>
#include <time.h>

using namespace std;


__global__
void reduction_kernel(int *d_a, int expo){

	int tid = blockIdx.x * blockDim.x + threadIdx.x ;

	int i = tid * pow(2, expo) ; 
	int next = i + pow(2, expo) / 2 ; 

	if(d_a[i] > d_a[next]){ // Doing Min Reduction swap

		int temp = d_a[i] ; 
		d_a[i] = d_a[next] ; 
		d_a[next] = temp ; 

	}	
	 // printf("%d\n", i);
}

void printArr(int *a, int N){
	cout << "Array is" << endl ; 
	for (int i = 0; i < N; i++)
	{
		cout << a[i] << "\t" ;
	}
  	cout << endl ; 	
}

int trad_min(int *a, int N){
	int mini = a[0] ; 
	for (int i = 0; i < N; i++)
	{
		if(a[i] < mini){
			mini = a[i] ; 
		}
	}
	return mini ; 
}

int foo(){

	int N = 1024 ; 
	int a[N] ; 

	for (int i = 0; i < N; i++)
	{
		a[i] = rand() ; 
	}

	int t_min = trad_min(a, N) ; 
	cout << "\nMinimum Element is " << t_min << endl; 
  	// printArr(a, N) ; 

	int *d_a;
	int bytes = sizeof(int) * N ; 

	hipMalloc(&d_a, bytes) ; 
	hipMemcpy(d_a, a, bytes, hipMemcpyHostToDevice) ; 

	int num_thread = N ; 
	int expo = 1 ; 
	while(num_thread != 0){
		num_thread = num_thread / 2 ; 
		reduction_kernel<<<1,num_thread>>>(d_a, expo) ;
		expo++ ;  

	}
	// reduction_kernel<<<1,N/2>>>(d_a) ; 

	hipMemcpy(a, d_a, bytes, hipMemcpyDeviceToHost) ; 

	hipFree(d_a) ; 

  	// printArr(a, N) ;
  	int c_min = a[0] ; 
  	cout << "\nMinimum Element is " << c_min << endl ;

  	return t_min == c_min ; 

}

int main(){
  srand(time(0)) ; 
  
   	for (int i = 0; i < 100; i++)
   	{
   		cout << foo() ; 
   	}
	
	return 0 ; 
}