#include "hip/hip_runtime.h"
%%cu
#include<iostream>
#include<cstdio>
#include <vector> 

using namespace std;


__global__
void vector_add_kernel(int *d_a, int *d_b, int *d_c){

  
	int id = blockIdx.x * blockDim.x + threadIdx.x ; 
  printf("%d \n", id);
	d_c[id] = d_a[id] + d_b[id] ; 

}

void trad_vector_add(int *a, int *b, int *c, int N){
	hipEvent_t start, end;
	float elapsed_time = 0.0f; 
	hipEventCreate(&start);
	hipEventCreate(&end);
	hipEventRecord(start) ; 

	for (int i = 0; i < N; i++)
	{
		c[i] = a[i] + b[i] ; 
	}

	hipEventRecord(end);
	hipEventSynchronize(end) ; 

	hipEventElapsedTime(&elapsed_time, start, end) ; 

	cout << "Traditional Time Required was " << elapsed_time << endl ; 

	for (int i = 0; i < 10; i++){
		
		cout << c[i] << " " << endl ; 

	}

}

int main(){

	int N = 10;
	int a[N], b[N], c[N]; 

	for(int i = 0 ; i < N ; i++){
		a[i] = i ; 
		b[i] = i ; 
	}

	int *d_a, *d_b, *d_c ; 

	int bytes = sizeof(int) * N ; 

	hipMalloc(&d_a, bytes) ; 
	hipMemcpy(d_a, a, bytes, hipMemcpyHostToDevice) ; 

	hipMalloc(&d_b, bytes) ; 
	hipMemcpy(d_b, b, bytes, hipMemcpyHostToDevice) ; 

	hipMalloc(&d_c, bytes) ; 

	// Timing Code

	hipEvent_t start, end;
	float elapsed_time = 0.0f; 
	hipEventCreate(&start);
	hipEventCreate(&end);
	hipEventRecord(start) ; 

	vector_add_kernel<<<10,N/10>>>(d_a, d_b, d_c);

	hipEventRecord(end);
	hipEventSynchronize(end) ; 

	hipEventElapsedTime(&elapsed_time, start, end) ; 

	cout << "Cuda Time Required was " << elapsed_time << endl ; 

	hipMemcpy(c, d_c, bytes, hipMemcpyDeviceToHost) ; 

	// Free Device Memory
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	for (int i = 0; i < N; i++){
		
		cout << c[i] << " " << endl ; 

	}
    
  // Doing the traditional way

	// trad_vector_add(a, b, c, N) ; 


	return 0 ; 
}