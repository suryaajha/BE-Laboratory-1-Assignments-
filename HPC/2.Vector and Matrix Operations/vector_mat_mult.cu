#include "hip/hip_runtime.h"
%%cu
#include<iostream>
#include <cstdio>

using namespace std ;

void print_arr(int *a, int N){

	cout << "Array is " << endl ;

	for (int i = 0; i < N; i++)
	{
		cout << a[i] << "\t" ; 
	}
	cout << endl ; 
}

void print_mat(int *matrix, int N, int M){

	cout << "Matrix is " << endl ;

	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < M; j++)
		{
			cout << matrix[ i * M + j] <<  "\t" ; 
		}
		cout << endl ; 
	}
	cout << endl ;
}

void trad_vector_mat_mult(int *vec, int *matrix, int N, int M){
	int store[M] ; 

	// Timing Code
	hipEvent_t start, end ;
	float elapsed_time = 0.0f ; 

	hipEventCreate(&start) ;
	hipEventCreate(&end) ;

	hipEventRecord(start) ; 

	// end of timing code

	for (int j = 0; j < M; j++)
	{
		store[j] = 0 ; 
		for (int i = 0; i < N; i++)
		{
			store[j] += vec[i] * matrix[i * M + j] ; 
		}
	}
	// timing code
	hipEventRecord(end) ; 
	hipEventSynchronize(end) ; 

	hipEventElapsedTime(&elapsed_time, start, end) ; 
	cout << "Sequential Version Vector Matrix Multiplication Time is " << elapsed_time << endl ; 

	// end of timing code
	print_arr(store, M) ; 
}

// Kernels

__global__
void vector_matrix_multiplication_kernel(int *d_vec, int *d_matrix, int *store, int N, int M){
	int tid = blockIdx.x * blockDim.x + threadIdx.x ; 
	int value = 0 ;

	for (int i = 0; i < N; i++)
	{
		value += d_vec[i] * d_matrix[i * M + tid] ;  
	}

	store[tid] = value ; 

}

// End of Kernels


int main(){
	int N = 10 ; 
	int vec[N], temp_store[N]; 

	for (int i = 0; i < N; i++)
	{
		vec[i] = i + 1 ; 
	}

	// print_arr(vec, N) ; 

	int M = N ; 
	int matrix[N * M] ; 
	int k = 1 ; 

	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < M; j++)
		{
			matrix[i * M + j] = k++ ; 
		}
	}

	// print_mat(matrix, N, M) ; 

	trad_vector_mat_mult(vec, matrix, N, M) ; 

	int *d_vec, *d_temp_store, *d_matrix ; 

	int vec_bytes = sizeof(int) * N ; 
	int mat_bytes = sizeof(int) * N * sizeof(int) * M ; 

	hipMalloc(&d_vec, vec_bytes) ; 
	hipMalloc(&d_temp_store, vec_bytes) ;
	hipMalloc(&d_matrix, mat_bytes) ;

	hipMemcpy(d_vec, vec, vec_bytes, hipMemcpyHostToDevice) ;  
	hipMemcpy(d_matrix, matrix, mat_bytes, hipMemcpyHostToDevice) ;  

	// Timing Code
	// hipEvent_t start, end ;
	// float elapsed_time = 0.0f ; 

	// hipEventCreate(&start) ;
	// hipEventCreate(&end) ;

	// hipEventRecord(start) ; 

	// end of timing code

	vector_matrix_multiplication_kernel<<<1, M>>>(d_vec, d_matrix, d_temp_store, N, M) ; 

	// timing code
	// hipEventRecord(end) ; 
	// hipEventSynchronize(end) ; 

	// hipEventElapsedTime(&elapsed_time, start, end) ; 
	// cout << "CUDA Version Vector Matrix Multiplication Time is " << elapsed_time << endl ; 
	// end of timing code

	hipMemcpy(temp_store, d_temp_store, vec_bytes, hipMemcpyDeviceToHost) ; 

	hipFree(d_vec);
	hipFree(d_matrix);
	hipFree(d_temp_store);


	print_arr(temp_store, N) ; 
	return  0 ; 
}